/*******************************************************************************
All of the GPU related code are here.
We will compile the code as a .so, and then link the code in the hashpipe code.
 ******************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include "hipfft/hipfft.h"
#include "hip/hip_runtime.h"

//extern "C" {
#include "fast_gpu.h"
// This is the PFB FIR code from James
#include "pfb_fir.cuh"

// The following buffers are the GPU buffers.
char            *data_in_gpu;       // input data
float           *weights_gpu;       // PFB FIR weights
hipfftReal       *pfbfir_out_gpu;    // the output of PFB FIR
hipfftComplex    *data_out_gpu;      // output data on GPU
hipfftComplex    *data_out_host;     // output data on host

// cufft plan
hipfftHandle plan;

// PFB FIR parameters
int step        = CHANNELS;
int out_n       = step * SPECTRA;
int stepy       = out_n/(256*1024)*step;
int groupsx     = step/WGS;
int groupsy     = (out_n + stepy - 1)/stepy;
dim3 dimgrid(groupsx*WGS, groupsy);
dim3 dimblock(WGS,1);

int GPU_GetDevInfo()
{
    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);
    printf("GPU Device Info:\r\n");
    printf("%-25s: %d\r\n", "MaxThreadsPerBlock", prop.maxThreadsPerBlock);
    printf("%-25s: %d %d %d\r\n","maxThreadsDim", prop.maxThreadsDim[0], \
                                                  prop.maxThreadsDim[1], \
                                                  prop.maxThreadsDim[2]);
    printf("%-25s: %d %d %d\r\n","maxGridSize",   prop.maxGridSize[0], \
                                                  prop.maxGridSize[1], \
                                                  prop.maxGridSize[2]);

    if(!prop.deviceOverlap)
        return -1;
    else
        return 0;
}

// This func is used for allocating pinned memory on the host computer 
//int Host_MallocBuffer(DIN_TYPE *buf_in, DOUT_TYPE *buf_out)
int Host_MallocBuffer(DIN_TYPE **buf_in, DOUT_TYPE **buf_out)
{
    hipError_t status;
    status = hipHostMalloc((void **)buf_in, SAMPLES * sizeof(DIN_TYPE));
    if(status != hipSuccess)
        return -1;
    status = hipHostMalloc((void **)buf_out, OUTPUT_LEN * sizeof(DOUT_TYPE));
    if(status != hipSuccess)
        return -2;
    return 0;
}

// This func is used for allocating memory on the GPU
void GPU_MallocBuffer()
{
    hipMalloc((void**)&data_in_gpu, SAMPLES * sizeof(char));
    hipMalloc((void**)&weights_gpu, TAPS*CHANNELS*sizeof(float));
    hipMalloc((void**)&pfbfir_out_gpu, CHANNELS*SPECTRA*sizeof(hipfftReal));
    hipMalloc((void**)&data_out_gpu, CHANNELS*SPECTRA * sizeof(hipfftComplex));
    hipHostMalloc((void**)&data_out_host, OUTPUT_LEN * sizeof(hipfftComplex));
}

// This func is used for creating cufft plan
int GPU_CreateFFTPlan()
{
    int rank = 1;
    int n[1];
    n[0] = CHANNELS;
    int istride = 1;
    int idist = CHANNELS;
    int ostride = 1;
    int odist = CHANNELS;
    
    int inembed[1], onembed[1];
    inembed[0] = CHANNELS*SPECTRA;
    onembed[0] = CHANNELS*SPECTRA;
    hipfftResult fft_ret = hipfftPlanMany(&plan, rank, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_R2C, SPECTRA);
    if( fft_ret != HIPFFT_SUCCESS )
        return -1;
    else
        return 0;
}

// move weights from host to GPU
void GPU_MoveWeightsFromHost(float *weights)
{
    hipMemcpy(weights_gpu, weights, TAPS * CHANNELS * sizeof(float), hipMemcpyHostToDevice);
}

// move data from host to GPU
void GPU_MoveDataFromHost(DIN_TYPE *din)
{
    hipMemcpy(data_in_gpu, din, SAMPLES * sizeof(DIN_TYPE), hipMemcpyHostToDevice);
}

// move data from GPU to host
void GPU_MoveDataToHost(DOUT_TYPE *dout)
{
//#pragma unroll 
    for(int i = 0; i < SPECTRA; i++)
    //hipMemcpy(dout + i * CH_PER_SPEC, data_out_gpu + i * CHANNELS + START_BIN, OUTPUT_LEN * sizeof(DOUT_TYPE), hipMemcpyDeviceToHost);
        hipMemcpy(data_out_host + i * CH_PER_SPEC, \
                   data_out_gpu + i * CHANNELS + START_BIN, 
                   OUTPUT_LEN * sizeof(hipfftComplex), 
                   hipMemcpyDeviceToHost);
    for(int i = 0; i < OUTPUT_LEN; i++)
        dout[i] = data_out_host[i].x * data_out_host[i].x + \
                  data_out_host[i].y * data_out_host[i].y;

}

// do PFB
int GPU_DoPFB()
{

    pfb_fir<<<dimgrid,dimblock>>>(
        (float *)pfbfir_out_gpu,  
        (char*)data_in_gpu,   
        weights_gpu,    
        out_n,
        step,
        stepy,
        0,
        0
        ); 
    hipDeviceSynchronize();
    hipfftResult fft_ret;
    fft_ret = hipfftExecR2C(plan, (hipfftReal*)pfbfir_out_gpu, (hipfftComplex*) data_out_gpu);
    if (fft_ret != HIPFFT_SUCCESS)
        return -1; 
    else
        return 0;
}

void GPU_DestroyPlan()
{
    hipfftDestroy(plan);
}

void Host_FreeBuffer(DIN_TYPE *buf_in, DOUT_TYPE *buf_out)
{
    hipHostFree(buf_in);
    hipHostFree(buf_out);
}

void GPU_FreeBuffer()
{
    hipFree(data_in_gpu);
    hipFree(weights_gpu);
    hipFree(pfbfir_out_gpu);
    hipFree(data_out_gpu);
    hipHostFree(data_out_host);
}
//}